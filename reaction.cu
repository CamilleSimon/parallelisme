#include "hip/hip_runtime.h"
// Travail réalisé en binôme par Corentin Ballot et Camille Simon
//
// Pour le bien de vos yeux, il est fortement conseillé de modifier
// les couleurs avant exécution
//
// Pour chaque jeu de vitesse de diffusion nous avons prévu un
// jeu de couleur assorti
// Retirez les commantaires des vitesses de diffusion pour découvrir
// un affichage pensé rien que pour vous

#include "gpu_bitmap.h"

#define WIDTH 800
#define HEIGHT 600
#define DIM 16

#define TAUX_REACTION_A 0.04f
#define TAUX_REACTION_I (TAUX_REACTION_A / 200)

#define TAUX_RESORPTION_A 0.06f
#define TAUX_RESORPTION_I TAUX_RESORPTION_A

// Lampe à lave
//#define VITESSE_DIFFUSION_A 50
//#define VITESSE_DIFFUSION_I 55

// Léopard revisité
//#define VITESSE_DIFFUSION_A 2
//#define VITESSE_DIFFUSION_I 22

// Années yeahyeah
#define VITESSE_DIFFUSION_A 5
#define VITESSE_DIFFUSION_I 10

#define TAUX_DIFFUSION_A 0.065f
#define TAUX_DIFFUSION_I 0.04f

#define SEUIL 130


__global__ void color(float *t, uchar4 *buf) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		float t3 = t[offset];
		float r, g, b;

		switch(VITESSE_DIFFUSION_A){
			case 5 :
				if (t3 < SEUIL) {
					r = 255; g = 0; b = 127;
				} else {
					r = 0; g = 47;	b = 167;
				}
				break;
			case 2 :
				if (t3 < SEUIL) {
					r = 240; g = 195; b = 0;
				} else {
					r = 63; g = 34;	b = 4;
				}
				break;
			case 50 :
				if (t3 < SEUIL) {
					r = 128; g = 0; b = 128;
				} else {
					r = 223; g = 109;	b = 20;
				}
				break;
		}

		buf[offset].x = r;
		buf[offset].y = g;
		buf[offset].z = b;
		buf[offset].w = 255;
	}
}

__global__ void reaction(float *a, float *a1, float *i, float *i1) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;

		// A est catalysé par A et inhibé par I
		a1[offset] = a[offset] + TAUX_REACTION_A * a[offset] * a[offset] / i[offset];
		// I est catalysé par A
		i1[offset] = i[offset] + TAUX_REACTION_I * a[offset] * a[offset];
		// la réaction consomme une certaine quantité de A et de I
		a1[offset] = (1 - TAUX_RESORPTION_A) * a1[offset];
		i1[offset] = (1 - TAUX_RESORPTION_I) * i1[offset];
	}
}

__global__ void diffusion(float *grille, float taux_diffusion) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < WIDTH && y < HEIGHT) {
		int offset = y * WIDTH + x;
		int top = y == HEIGHT - 1 ? offset : offset + WIDTH;
		int bottom = y == 0 ? offset : offset - WIDTH;
		int left = x == 0 ? offset : offset - 1;
		int right = x == WIDTH - 1 ? offset : offset + 1;

		grille[offset] = (1 - 4 * taux_diffusion) * grille[offset] +
		taux_diffusion * (grille[top] + grille[bottom] + grille[left] + grille[right]);
	}
}

struct Data {
	float *A;
	float *A_next;
	float *I;
	float *I_next;
	dim3 blocks;
	dim3 threads;
};

void render_callback(uchar4 *buf, Data *data, int ticks) {
	reaction<<<data->blocks, data->threads>>>(data->A, data->A_next, data->I, data->I_next);
	reaction<<<data->blocks, data->threads>>>(data->A_next, data->A, data->I_next, data->I);

	for (int s = 0; s < VITESSE_DIFFUSION_A; s++)
		diffusion<<<data->blocks, data->threads>>>(data->A, TAUX_DIFFUSION_A);
	for (int s = 0; s < VITESSE_DIFFUSION_I; s++)
		diffusion<<<data->blocks, data->threads>>>(data->I, TAUX_DIFFUSION_I);

	color<<<data->blocks, data->threads>>>(data->A_next, buf);
}

void clean_callback(Data *data) {
	HANDLE_CUDA_ERR(hipFree(data->A));
	HANDLE_CUDA_ERR(hipFree(data->I));
}

int main() {
	Data data;
	GPUBitmap bitmap(WIDTH, HEIGHT, &data, "Heat");

	size_t size = WIDTH * HEIGHT * sizeof(float);

	float *a_initial = (float *)calloc(WIDTH * HEIGHT, sizeof(float));
	float *i_initial = (float *)calloc(WIDTH * HEIGHT, sizeof(float));

	for (int y = 0; y <  HEIGHT * WIDTH + WIDTH; y++) {
			a_initial[y] = (rand() % 100) + 1;
			i_initial[y] = (rand() % 100) + 1;
	}

	data.blocks = dim3((WIDTH + DIM - 1) / DIM, (HEIGHT + DIM - 1) / DIM);
	data.threads = dim3(DIM, DIM);
	HANDLE_CUDA_ERR(hipMalloc(&data.A, size));
	HANDLE_CUDA_ERR(hipMalloc(&data.I, size));
	HANDLE_CUDA_ERR(hipMalloc(&data.A_next, size));
	HANDLE_CUDA_ERR(hipMalloc(&data.I_next, size));
	HANDLE_CUDA_ERR(hipMemcpy(data.A, a_initial, size, hipMemcpyHostToDevice));
	HANDLE_CUDA_ERR(hipMemcpy(data.I, i_initial, size, hipMemcpyHostToDevice));

	bitmap.animate((void (*)(uchar4*, void*, int))render_callback, (void (*)(void*))clean_callback);
	return 0;
}